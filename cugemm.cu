#include "hip/hip_runtime.h"
/*TODO: before you submit on Canvas, include here:
     1) GPU Using: GeForce RTX 2060
     2) Final performance: Average elapsed time: (0.050167) s, performance: ( 342.45) GFLOPS. size: (2048).
        In Homework 1, even though I fixed coalesced memory access, the performance was still low. I followed the
        instruction in transpose.cu and used a shared memory bank but still the performance was lower than expected in 
        homework 1 hint. Here are the specification of my GPU:
        
            Name: NVIDIA GeForce RTX 2060
            Compute capability: 7.5
            MultiProcessor (SM) count: 30
            Warp size: 32
            Max threads per block: 1024
            Max threads per SM: 1024
            Max threads dim: (1024, 1024, 64)
            Max grid size:   (2147483647, 65535, 65535)

        From Nsight Compute, I located the Global Load & Store Sectors/Request (ld) for coalesced implementation and the value is 4.0, in comparison
        the value for uncoalesced implementation is 16.5. So I believe the memory access is coalesced.
*/

#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <random>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// from https://github.com/jarro2783/cxxopts
#include "cxxopts.hpp"

#define cudaCheck(err) (cudaErrorCheck(err, __FILE__, __LINE__))
#define cublasCheck(err) (cublasErrorCheck(err, __FILE__, __LINE__))
#define ROUND_UP_TO_NEAREST(M, N) (((M) + (N)-1) / (N))

enum Algo
{
    cublas = 0,
    basic,
    gmem_coalesced,
    smem,
    smem_multioutput,
    numAlgos
};

const char *algo2str(Algo a)
{
    switch (a)
    {
    case cublas:
        return "cublas";
    case basic:
        return "basic";
    case gmem_coalesced:
        return "gmem_coalesced";
    case smem:
        return "sharedmem";
    case smem_multioutput:
        return "sharedmem_multioutput";
    default:
        return "INVALID";
    }
}

void cudaErrorCheck(hipError_t error, const char *file, int line);
void cublasErrorCheck(hipblasStatus_t status, const char *file, int line);
void randomize_matrix(float *mat, int N);
void const_init_matrix(float *mat, int N, float F);
bool verify_matrix(float *expected, float *actual, int M, int N);
void print_matrix(const float *A, int M, int N, std::ostream &outs);
void runAlgo(Algo algo, hipblasHandle_t handle, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);
void runCublas(hipblasHandle_t handle, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);

const std::string errLogFile = "gemmValidationFailure.txt";

// NB: must use a single generator to avoid duplicates
std::default_random_engine generator(2);
std::uniform_real_distribution<float> distribution(0, 1);

// Variables defined while completing homework questions
const int TILE_N     = 64;  // columns per block
const int BLOCK_ROWS = 16;   // rows per block

int main(int argc, char **argv)
{
    // command-line flags
    cxxopts::Options options("gemm.cu", "CUDA GEMM kernels");
    options.add_options()("size", "matrix size (N x N)", cxxopts::value<uint16_t>()->default_value("128"))                //
        ("reps", "repeat GEMM this many times", cxxopts::value<uint16_t>()->default_value("1"))                           //
        ("algo", "GEMM algorithm to use, a number in [0,4], 0 is cuBLAS", cxxopts::value<uint16_t>()->default_value("0")) //
        ("validate", "Validate output against cuBLAS", cxxopts::value<bool>()->default_value("true"))                     //
        ("rngseed", "PRNG seed", cxxopts::value<uint>()->default_value("2"))                     //
        ("h,help", "Print usage");

    auto clFlags = options.parse(argc, argv);
    if (clFlags.count("help"))
    {
        std::cout << options.help() << std::endl;
        exit(0);
    }
    const uint16_t SIZE = clFlags["size"].as<uint16_t>();
    if (SIZE % 32 != 0)
    {
        //std::cout << "--size must be a multiple of 32" << std::endl;
        //exit(EXIT_FAILURE);
    }
    const uint16_t REPS = clFlags["reps"].as<uint16_t>();
    const Algo ALGO = static_cast<Algo>(clFlags["algo"].as<uint16_t>());
    if (ALGO >= numAlgos)
    {
        printf("Invalid algorithm: %d\n", ALGO);
        exit(EXIT_FAILURE);
    }

    const bool VALIDATE = clFlags["validate"].as<bool>();
    const uint SEED = clFlags["rngseed"].as<uint>();
    generator.seed(SEED);
    printf("Multiplying two %u x %u matrices with %u trials using %s algorithm\n", SIZE, SIZE, REPS, algo2str(ALGO));

    cudaCheck(hipSetDevice(0));

    // Setup cublas
    hipblasHandle_t handle;
    cublasCheck(hipblasCreate(&handle));

    // Using cudaEvent for gpu stream timing, cudaEvent is equivalent to
    // publishing event tasks in the target stream
    hipEvent_t beg, end;
    cudaCheck(hipEventCreate(&beg));
    cudaCheck(hipEventCreate(&end));

    uint16_t m = SIZE, n = SIZE, k = SIZE;

    // GEMM computes C = α*AB+β*C

    // just do pure A*B (for simpler debugging)
    float alpha = 1.0, beta = 1.0, initC = 1.0;

    float *A = nullptr, *B = nullptr, *C = nullptr, *C_ref = nullptr;     // host matrices
    float *dA = nullptr, *dB = nullptr, *dC = nullptr, *dC_ref = nullptr; // device matrices

    A = (float *)malloc(sizeof(float) * SIZE * SIZE);
    B = (float *)malloc(sizeof(float) * SIZE * SIZE);
    C = (float *)malloc(sizeof(float) * SIZE * SIZE);
    C_ref = (float *)malloc(sizeof(float) * SIZE * SIZE);

    randomize_matrix(A, SIZE * SIZE);
    randomize_matrix(B, SIZE * SIZE);
    randomize_matrix(C, SIZE * SIZE);

    const_init_matrix(C, SIZE * SIZE, initC);
    // print_matrix(A, SIZE, SIZE, std::cout);
    // print_matrix(B, SIZE, SIZE, std::cout);
    // print_matrix(C, SIZE, SIZE, std::cout);

    cudaCheck(hipMalloc((void **)&dA, sizeof(float) * SIZE * SIZE));
    cudaCheck(hipMalloc((void **)&dB, sizeof(float) * SIZE * SIZE));
    cudaCheck(hipMalloc((void **)&dC, sizeof(float) * SIZE * SIZE));
    cudaCheck(hipMalloc((void **)&dC_ref, sizeof(float) * SIZE * SIZE));

    cudaCheck(hipMemcpy(dA, A, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dB, B, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC, C, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC_ref, C, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice));

    printf("dimensions(m=n=k) %u, alpha: %f, beta: %f\n", m, alpha, beta);

    // Verify the correctness of the calculation, and execute it once before the
    // kernel function timing to avoid cold start errors
    if (!VALIDATE)
    {
        printf("disabled validation\n");
    }
    else
    {
        // run cublas to get correct answer in dC_ref
        runCublas(handle, m, n, k, alpha, dA, dB, beta, dC_ref);

        // run user's algorithm, filling in dC
        runAlgo(ALGO, handle, m, n, k, alpha, dA, dB, beta, dC);

        cudaCheck(hipDeviceSynchronize());

        // copy both results back to host
        hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);
        hipMemcpy(C_ref, dC_ref, sizeof(float) * m * n, hipMemcpyDeviceToHost);

        if (verify_matrix(C_ref, C, n, m))
        {
            printf("Validated successfully!\n");
        }
        else
        {
            printf("Failed validation against NVIDIA cuBLAS.\n");
            std::cout << " Logging faulty output into " << errLogFile << "\n";
            std::ofstream fs;
            fs.open(errLogFile, std::ios::out | std::ios::trunc);
            fs << "α=" << alpha << " β=" << beta << std::endl;
            fs << "C matrix initialized to " << initC << std::endl << std::endl;
            fs << "A:" << std::endl;
            print_matrix(A, m, n, fs);
            fs << "B:" << std::endl;
            print_matrix(B, m, n, fs);
            fs << "C:" << std::endl;
            print_matrix(C, m, n, fs);
            fs << "Expected:" << std::endl;
            print_matrix(C_ref, m, n, fs);
            fs.close();
            exit(EXIT_FAILURE);
        }
    }

    // timing run(s)
    hipEventRecord(beg);
    for (int j = 0; j < REPS; j++)
    {
        // We don't reset dC between runs to save time
        runAlgo(ALGO, handle, m, n, k, alpha, dA, dB, beta, dC);
        cudaCheck(hipDeviceSynchronize());
    }

    // TODO: measure timing without memory transfers?
    cudaCheck(hipEventRecord(end));
    cudaCheck(hipEventSynchronize(beg));
    cudaCheck(hipEventSynchronize(end));
    float elapsed_time;
    cudaCheck(hipEventElapsedTime(&elapsed_time, beg, end));
    elapsed_time /= 1000.; // Convert to seconds

    double flops = (double)2 * m * n * k;
    printf(
        "Average elapsed time: (%7.6f) s, performance: (%7.2f) GFLOPS. size: (%u).\n",
        elapsed_time / REPS,
        (REPS * flops * 1e-9) / elapsed_time,
        m);

    // free CPU and GPU memory
    free(A);
    free(B);
    free(C);
    free(C_ref);
    cudaCheck(hipFree(dA));
    cudaCheck(hipFree(dB));
    cudaCheck(hipFree(dC));
    cudaCheck(hipFree(dC_ref));
    cublasCheck(hipblasDestroy(handle));

    return 0;
}

/** Function to check for errors in CUDA API calls */
void cudaErrorCheck(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        printf("[CUDA ERROR] at file %s:%d:\n%s: %s\n", file, line,
               hipGetErrorName(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};

void cublasErrorCheck(hipblasStatus_t status, const char *file, int line)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("[CUDA ERROR] at file %s:%d:\n %s: %s\n", file, line,
               cublasGetStatusName(status), cublasGetStatusString(status));
        exit(EXIT_FAILURE);
    }
}

/** Initialize the given matrix `mat` which has `N` contiguous values. Contents of `mat` are set to random values. */
void randomize_matrix(float *mat, int N)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = distribution(generator);
    }
}

void const_init_matrix(float *mat, int N, float F)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = F;
    }
}

/** Print the given MxN matrix `mat` to the provided output stream. */
void print_matrix(const float *A, int M, int N, std::ostream &outs)
{
    outs << "[";
    for (int i = 0; i < M * N; i++)
    {
        if ((i + 1) % N == 0)
        {
            outs << std::fixed << std::setprecision(3) << A[i];
        }
        else
        {
            outs << std::fixed << std::setprecision(3) << A[i] << ", ";
        }
        if ((i + 1) % N == 0)
        {
            if (i + 1 < M * N)
                outs << ";" << std::endl;
        }
    }
    outs << "]" << std::endl << std::endl;
}

bool verify_matrix(float *expected, float *actual, int M, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < M; j++)
        {
            float fexp = (expected[(i * N) + j]);
            float fact = (actual[(i * N) + j]);
            double diff = std::fabs(fexp - fact);
            if (diff > 0.002)
            {
                printf("Divergence! Should be %5.3f, is %5.3f (diff %5.3f) at [%d,%d]\n",
                       fexp, fact, diff, i, j);
                return false;
            }
        }
    }
    return true;
}

void runCublas(hipblasHandle_t handle, int M, int N, int K, float alpha,
               float *A, float *B, float beta, float *C)
{
    // cuBLAS uses *column-major* order. So we change the order of our row-major A &
    // B, since (B^T*A^T)^T = (A*B)
    // hipblasStatus_t ok = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_16F,
    //                                  N, A, HIP_R_16F, K, &beta, C, HIP_R_16F, N, /*HIPBLAS_COMPUTE_16F*/ HIPBLAS_COMPUTE_16F_PEDANTIC,
    //                                  HIPBLAS_GEMM_DEFAULT);
    hipblasStatus_t ok = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, N, A, K, &beta, C, N);
    cublasCheck(ok);
}

__global__ void runBasic(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
    const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < M && y < N)
    {
        float tmp = 0.0;
        // C = α*(AxB)+β*C
        for (int i = 0; i < K; ++i)
        {
            // tmp += __A__[x][i] * __B__[i][y]
            tmp += A[(x * K) + i] * B[(i * N) + y];
        }
        // __C__[x][y]
        C[(x * N) + y] = (alpha * tmp) + (beta * C[x * N + y]);
    }
}

__global__ void runGmemCoalesced(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
    // HW1 TODO: copy runBasic() code here and update to avoid uncoalesced accesses to global memory.
    // Note, you are also free to change the grid dimensions in the kernel launch below.

    // Create a shared memory to hold input tiles
    __shared__ float As[BLOCK_ROWS][TILE_N + 1];
    __shared__ float Bs[TILE_N][TILE_N + 1];

    const int col = blockIdx.x * TILE_N     + threadIdx.x;
    const int row = blockIdx.y * BLOCK_ROWS + threadIdx.y;
    if (row >= M || col >= N) return;

    float acc = 0;

    // Tile K in chunks of TILE_N
    for (int k0 = 0; k0 < K; k0 += TILE_N) {
        // Threads to load multiple rows of A and B
        int aCol = k0 + threadIdx.x;
        if (aCol < K) {
            As[threadIdx.y][threadIdx.x] = A[row * K + aCol];
        } else {
            As[threadIdx.y][threadIdx.x] = 0;
        }

        for (int kk = threadIdx.y; kk < TILE_N; kk += BLOCK_ROWS) {
            int bRow = k0 + kk;
            if (bRow < K) {
                Bs[kk][threadIdx.x] = B[bRow * N + col];
            } else {
                Bs[kk][threadIdx.x] = 0;
            }
        }

        __syncthreads();

        // Compute partial product
        for (int k = 0; k < TILE_N; ++k) {
            acc += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write back
    const int idx = row * N + col;
    C[idx] = alpha * acc + beta * C[idx];
}

const uint F = 32;

__global__ void runSharedMem(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
    // HW2 TODO: Use shared memory to cache square FxF tiles of the A and B matrices in shared memory 
    // (SA and SB, respectively, provided below). Each thread should compute the result for one cell 
    // of the output matrix C.

    // Note, you will also need to change the grid dimensions in the kernel launch below to take into account the value
    // of F (which is a constant, defined above). You should experiment with different values of F to see how it 
    // affects performance.

    __shared__ float SA[F][F];
    __shared__ float SB[F][F];

}

const uint G = 4;

__global__ void runSharedMemMultiOutput(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
    // HW3 TODO: Copy your runSharedMem() code here and update it so that each thread computes the result for GxG cells 
    // of the output matrix C. Each thread should accumulate temporary results in the local LC matrix, provided below,
    // before writing them to C in global memory.

    // Note, you will also need to change the grid dimensions in the kernel launch below. You should experiment 
    // with different values of F and G to see how they affect performance.

    __shared__ float SA[F][F];
    __shared__ float SB[F][F];

    float LC[G][G] = {0.0};

}

void runAlgo(Algo algo, hipblasHandle_t handle, int M, int N, int K, float alpha,
             float *A, float *B, float beta, float *C)
{
    switch (algo)
    {
    case cublas:
        runCublas(handle, M, N, K, alpha, A, B, beta, C);
        break;
    case basic:
    {
        dim3 gridDim(ROUND_UP_TO_NEAREST(M, 32), ROUND_UP_TO_NEAREST(N, 32));
        dim3 blockDim(32, 32);
        runBasic<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
        break;
    }
    case gmem_coalesced:
    {
        dim3 threads(TILE_N, BLOCK_ROWS);
        dim3 blocks((N + TILE_N     - 1) / TILE_N,
                    (M + BLOCK_ROWS - 1) / BLOCK_ROWS);

        runGmemCoalesced<<<blocks, threads>>>(M, N, K, alpha, A, B, beta, C);
        break;
    }
    case smem:
    {
        assert(0 == M % F);
        assert(0 == N % F);
        assert(0 == K % F);
        // TODO: update your grid here
        dim3 gridDim(ROUND_UP_TO_NEAREST(M, 32), ROUND_UP_TO_NEAREST(N, 32));
        dim3 blockDim(32, 32);
        runSharedMem<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
        break;
    }
    case smem_multioutput:
    {
        assert(0 == M % F);
        assert(0 == N % F);
        assert(0 == K % F);
        assert(0 == F % G);
        assert((F*F) / (G*G) >= F);
        // TODO: update your grid here
        dim3 gridDim(ROUND_UP_TO_NEAREST(M, 32), ROUND_UP_TO_NEAREST(N, 32));
        dim3 blockDim(32, 32);
        runSharedMemMultiOutput<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
        break;
    }
    default:
        printf("Invalid algorithm: %d\n", algo);
        exit(EXIT_FAILURE);
    }
    cudaCheck(hipDeviceSynchronize()); // wait for kernel to finish
    cudaCheck(hipGetLastError());      // check for errors from kernel run
}
